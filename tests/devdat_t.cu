#include <devdat/devdat.hpp>

#include "catch2/catch.hpp"

TEST_CASE( "cuda_version", "[cuda]") {
    int driver_version , runtime_version;
    hipDriverGetVersion( &driver_version );
    hipRuntimeGetVersion ( &runtime_version );
    std::cout << driver_version << "\t" << runtime_version << std::endl;
}

TEST_CASE( "dev_dat", "[devdat]"){
    std::vector< std::vector<cudaT> > data{std::vector<cudaT> {0, 1, 2}, std::vector<cudaT> {2, 3, 4}};
}

// ToDo
/* TEST_CASE( "add_one", "[adder]" ){
  REQUIRE(add_one(0) == 1);
  REQUIRE(add_one(123) == 124);
  REQUIRE(add_one(-1) == 0);
} */